#include "hip/hip_runtime.h"
#include<stdio.h>


__global__ void hello_cu()
{
    printf("bid.x=%d, bid.y=%d, tid.x=%d\n", blockIdx.x, blockIdx.y, threadIdx.x);
}

__global__ void addArray(int *a, int *b, int *c)
{
    const int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < 5)
        c[id] = a[id] + b[id];
}

void gpuInfo()
{
    hipDeviceProp_t cudaProp;
    hipGetDeviceProperties(&cudaProp, 0);

    printf("device name:             %s\n", cudaProp.name);
    printf("compute capability       %d.%d\n", cudaProp.major, cudaProp.minor);
    printf("global memory:           %lu MB\n", cudaProp.totalGlobalMem >> 20);
    printf("constant memory:         %lu KB\n", cudaProp.totalConstMem >> 10);
    printf("grid size:               %d %d %d\n", cudaProp.maxGridSize[0],\
                            cudaProp.maxGridSize[1], cudaProp.maxGridSize[2]);
    printf("block size:              %d %d %d\n", cudaProp.maxThreadsDim[0],\
                            cudaProp.maxThreadsDim[1],cudaProp.maxThreadsDim[2]);
    printf("multiProcessorCount:        %d\n", cudaProp.multiProcessorCount);
    printf("maxThreadsPerMultiProcessor %d\n", cudaProp.maxThreadsPerMultiProcessor);
    printf("maxBlocksPerMultiProcessor  %d\n", cudaProp.maxBlocksPerMultiProcessor);
    printf("maxThreadsPerBlock:         %d\n", cudaProp.maxThreadsPerBlock);
    printf("accessPolicyMaxWindowSize %d\n", cudaProp.accessPolicyMaxWindowSize);
    printf("memoryBusWidth           %d\n", cudaProp.memoryBusWidth);
    printf("memoryClockRate          %d\n", cudaProp.memoryClockRate);
    printf("managedMemory            %d\n", cudaProp.managedMemory);
    printf("canMapHostMemory         %d\n", cudaProp.canMapHostMemory);
    printf("accessPolicyMaxWindowSize %d\n", cudaProp.accessPolicyMaxWindowSize);
    printf("asyncEngineCount         %d\n", cudaProp.asyncEngineCount);
    
    printf("FP64 support: %s, INT64 support: Yes\n", (cudaProp.major >= 2) ? "Yes (但性能低)" : "No");
}

void addTest()
{
    int a[] = {1,2,3,4,5};
    int *b, *c, *d;
    hipMalloc(&b, (size_t)20);
    hipMalloc(&c, (size_t)20);
    hipMalloc(&d, (size_t)20);
    hipMemset(b, 0, 20);
    hipMemcpy(b, a, 3*4, hipMemcpyHostToDevice);
    hipMemcpy(c, a, 5*4, hipMemcpyHostToDevice);

    dim3 block(32);
    dim3 grid(1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    addArray<<<block, grid>>>(b,c,d);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("using time %f ms\n", elapsedTime);

    hipMemcpy(a, d, 5*4, hipMemcpyDeviceToHost);

    for(int i=0; i<5; i++)
    {
        printf("a[%d] = %d\n", i, a[i]);
    }
    hipFree(b);
    hipFree(c);
    hipFree(d);
    hipDeviceReset();
}

int main(int argc, char const *argv[])
{
    hello_cu<<<2, 3>>>();
    printf("hello\n");
    hipDeviceSynchronize();

    int devices = 0;
    hipGetDeviceCount(&devices);
    printf("devices = %d\n", devices);
    hipError_t dick = hipSetDevice(0);   //调用gpu：0
    printf("hipSuccess = %d, dick = %d\n", hipSuccess, dick);

    gpuInfo();
    return 0;
}
