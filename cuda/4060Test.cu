// nvcc fc5_test.cu -lcublas -o fc5_test
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CHECK_CUDA(call)  { const hipError_t e = call; if (e != hipSuccess) { printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); exit(1); } }
#define CHECK_CUBLAS(call) { const hipblasStatus_t s = call; if (s != HIPBLAS_STATUS_SUCCESS) { printf("cuBLAS error %s:%d\n", __FILE__, __LINE__); exit(1); } }

const int batch_size = 64;
const int dim_in = 4096;
const int dim_hidden = 4096;
// const int dim_out = 4096;
const int layers = 5;

__global__ void relu(float* x, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) x[i] = fmaxf(0.0f, x[i]);
}

__global__ void relu_half(__half* x, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) x[i] = __hgt(x[i], __float2half(0.0f)) ? x[i] : __float2half(0.0f);
}

void fc5_cublas_fp32(hipblasHandle_t handle, float* input, float** weights, float** biases, float* output) {
    const float alpha = 1.0f, beta = 1.0f;
    float *x = input, *y;
    int M = batch_size, N = dim_hidden, K = dim_in;
    for (int l = 0; l < layers; l++) {
        y = output + l * M * N;  // 每层结果单独存
        CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, weights[l], N, x, K, &beta, y, N));
        CHECK_CUDA(hipMemcpy(y, biases[l], sizeof(float)*N, hipMemcpyDeviceToDevice));  // 加 bias
        int size = M * N;
        relu<<<(size+255)/256, 256>>>(y, size);
        x = y;
        K = N;
    }
}

void fc5_cublas_fp16(hipblasHandle_t handle, __half* input, __half** weights, __half** biases, __half* output) {
    const __half alpha = __float2half(1.0f), beta = __float2half(1.0f);
    __half *x = input, *y;
    int M = batch_size, N = dim_hidden, K = dim_in;
    for (int l = 0; l < layers; l++) {
        y = output + l * M * N;
        CHECK_CUBLAS(hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K,
            &alpha, weights[l], HIP_R_16F, N,
                    x, HIP_R_16F, K,
            &beta,  y, HIP_R_16F, N,
            HIPBLAS_COMPUTE_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
        CHECK_CUDA(hipMemcpy(y, biases[l], sizeof(__half)*N, hipMemcpyDeviceToDevice));
        int size = M * N;
        relu_half<<<(size+255)/256, 256>>>(y, size);
        x = y;
        K = N;
    }
}

int main() {
    printf("Allocating...\n");
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    size_t size_f32 = batch_size * dim_hidden * layers * sizeof(float);
    size_t size_f16 = batch_size * dim_hidden * layers * sizeof(__half);

    // FP32 alloc
    float *input32, *output32, *weights32[layers], *bias32[layers];
    CHECK_CUDA(hipMalloc(&input32, batch_size * dim_in * sizeof(float)));
    CHECK_CUDA(hipMalloc(&output32, size_f32));
    for (int l=0; l<layers; l++) {
        CHECK_CUDA(hipMalloc(&weights32[l], dim_hidden * dim_in * sizeof(float)));
        CHECK_CUDA(hipMalloc(&bias32[l], dim_hidden * sizeof(float)));
    }

    // FP16 alloc
    __half *input16, *output16, *weights16[layers], *bias16[layers];
    CHECK_CUDA(hipMalloc(&input16, batch_size * dim_in * sizeof(__half)));
    CHECK_CUDA(hipMalloc(&output16, size_f16));
    for (int l=0; l<layers; l++) {
        CHECK_CUDA(hipMalloc(&weights16[l], dim_hidden * dim_in * sizeof(__half)));
        CHECK_CUDA(hipMalloc(&bias16[l], dim_hidden * sizeof(__half)));
    }

    printf("Warmup...\n");
    fc5_cublas_fp32(handle, input32, weights32, bias32, output32);
    fc5_cublas_fp16(handle, input16, weights16, bias16, output16);
    CHECK_CUDA(hipDeviceSynchronize());

    printf("Measuring FP32...\n");
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start));
    for (int i=0; i<10; i++)
        fc5_cublas_fp32(handle, input32, weights32, bias32, output32);
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));
    float ms_fp32;
    hipEventElapsedTime(&ms_fp32, start, stop);

    printf("Measuring FP16 Tensor Core...\n");
    CHECK_CUDA(hipEventRecord(start));
    for (int i=0; i<10; i++)
        fc5_cublas_fp16(handle, input16, weights16, bias16, output16);
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));
    float ms_fp16;
    hipEventElapsedTime(&ms_fp16, start, stop);

    printf("FP32 time: %.3f ms\n", ms_fp32);
    printf("FP16 Tensor Core time: %.3f ms\n", ms_fp16);

    // Clean up
    CHECK_CUBLAS(hipblasDestroy(handle));
    hipFree(input32); hipFree(output32);
    hipFree(input16); hipFree(output16);
    for (int l=0; l<layers; l++) {
        hipFree(weights32[l]); hipFree(bias32[l]);
        hipFree(weights16[l]); hipFree(bias16[l]);
    }
    return 0;
}
